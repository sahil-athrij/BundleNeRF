#include "hip/hip_runtime.h"

#include "SIFTImageManager.h"
#include "cudaUtil.h"
#include "CUDATimer.h"

#define SORT_NUM_BLOCK_THREADS_X (MAX_MATCHES_PER_IMAGE_PAIR_RAW / 2)

int CheckErrorCUDA(const char* location)
{
#if (defined(_DEBUG) || defined(DEBUG))
	hipDeviceSynchronize();
	hipError_t e = hipGetLastError();
	if (e)
	{
		if (location) fprintf(stderr, "%s:\t", location);
		fprintf(stderr, "%s\n", hipGetErrorString(e));
		assert(0);
		return 1;
	}
	else
	{
		return 0;
	}
#else
	return 0;
#endif
}

__device__ bool cmpAndSawp(
	volatile float* dist0,
	volatile uint2* idx0,
	volatile float* dist1,
	volatile uint2* idx1
	)
{
	if (dist0[0] > dist1[0]) {
		float tmpDist = dist0[0];
		dist0[0] = dist1[0];
		dist1[0] = tmpDist;

		const unsigned int tmpIdxX = idx0[0].x;
		idx0[0].x = idx1[0].x;
		idx1[0].x = tmpIdxX;

		const unsigned int tmpIdxY = idx0[0].y;
		idx0[0].y = idx1[0].y;
		idx1[0].y = tmpIdxY;
		return true;
	}
	else {
		return false;
	}
}

#define FILTER_NUM_BLOCK_THREADS_X MAX_MATCHES_PER_IMAGE_PAIR_RAW



#define FILTER_DENSE_VERIFY_THREAD_SPLIT 32

#ifdef CUDACACHE_FLOAT_NORMALS
__device__ float3 computeProjError(unsigned int idx, unsigned int imageWidth, unsigned int imageHeight,
	float distThresh, float normalThresh, float colorThresh, const float4x4& transform, const float4x4& intrinsics,
	const float* d_inputDepth, const float4* d_inputCamPos, const float4* d_inputNormal, const float* d_inputColor,
	const float* d_modelDepth, const float4* d_modelCamPos, const float4* d_modelNormal, const float* d_modelColor,
	float sensorDepthMin, float sensorDepthMax)
#elif defined(CUDACACHE_UCHAR_NORMALS)
__device__ float3 computeProjError(unsigned int idx, unsigned int imageWidth, unsigned int imageHeight,
	float distThresh, float normalThresh, float colorThresh, const float4x4& transform, const float4x4& intrinsics,
	const float* d_inputDepth, const float4* d_inputCamPos, const uchar4* d_inputNormal, const float* d_inputColor,
	const float* d_modelDepth, const float4* d_modelCamPos, const uchar4* d_modelNormal, const float* d_modelColor,
	float sensorDepthMin, float sensorDepthMax)
#endif
{
	float3 out = make_float3(0.0f);

	float4 pInput = d_inputCamPos[idx];
#ifdef CUDACACHE_FLOAT_NORMALS
	float4 nInput = d_inputNormal[idx]; nInput.w = 0.0f;
#else
	float4 nInput = make_float4(MINF);
	uchar4 nInputU4 = d_inputNormal[idx];
	if (*(int*)(&nInputU4) != 0) nInput = make_float4(make_float3(nInputU4.x, nInputU4.y, nInputU4.z) / 255.0f * 2.0f - 1.0f, 0.0f);
#endif
	float dInput = d_inputDepth[idx];

	if (pInput.x != MINF && nInput.x != MINF && dInput >= sensorDepthMin && dInput <= sensorDepthMax) {
		const float4 pTransInput = transform * pInput;
		const float4 nTransInput = transform * nInput;

		float3 tmp = intrinsics * make_float3(pTransInput.x, pTransInput.y, pTransInput.z);
		const int2 screenPos = make_int2((int)roundf(tmp.x / tmp.z), (int)roundf(tmp.y / tmp.z));

		if (screenPos.x >= 0 && screenPos.y >= 0 && screenPos.x < (int)imageWidth && screenPos.y < (int)imageHeight) {
			float4 pTarget = d_modelCamPos[screenPos.y * imageWidth + screenPos.x];
#ifdef CUDACACHE_FLOAT_NORMALS
			float4 nTarget = d_modelNormal[screenPos.y * imageWidth + screenPos.x];
#else
			float4 nTarget = make_float4(MINF);
			uchar4 nTargetU4 = d_modelNormal[idx];
			if (*(int*)(&nTargetU4) != 0) nTarget = make_float4(make_float3(nTargetU4.x, nTargetU4.y, nTargetU4.z) / 255.0f * 2.0f - 1.0f, 0.0f);
#endif
			if (pTarget.x != MINF && nTarget.x != MINF) {
				float d = length(pTransInput - pTarget);
				float dNormal = dot(make_float3(nTransInput.x, nTransInput.y, nTransInput.z), make_float3(nTarget.x, nTarget.y, nTarget.z));
				float projInputDepth = pTransInput.z;
				float tgtDepth = d_modelDepth[screenPos.y * imageWidth + screenPos.x];

				if (tgtDepth >= sensorDepthMin && tgtDepth <= sensorDepthMax) {
					bool b = ((tgtDepth != MINF && projInputDepth < tgtDepth) && d > distThresh);
					if ((dNormal >= normalThresh && d <= distThresh /*&& c <= colorThresh*/) || b) {

						const float cameraToKinectProjZ = (pTransInput.z - sensorDepthMin) / (sensorDepthMax - sensorDepthMin);
						const float weight = max(0.0f, 0.5f*((1.0f - d / distThresh) + (1.0f - cameraToKinectProjZ)));

						out.x = d;
						out.y = weight;
						out.z = 1.0f;
					}
				}
			}
		}
	}

	return out;
}


void __global__ AddCurrToResidualsCU_Kernel(
	unsigned int curFrame,
	unsigned int startFrame,
	EntryJ* d_globMatches,
	uint2* d_globMatchesKeyPointIndices,
	int* d_globNumImagePairs,
	const int* d_currNumFilteredMatchesPerImagePair,
	const uint2* d_currFilteredMatchKeyPointIndices,
	const SIFTKeyPoint* d_keyPoints,
	const unsigned int maxKeyPointsPerImage,
	const float4x4 colorIntrinsicsInv
	)
{
	const unsigned int imagePairIdx = blockIdx.x + startFrame;
	if (imagePairIdx == curFrame) return;
	const unsigned int tidx = threadIdx.x;
	const unsigned int numMatches = d_currNumFilteredMatchesPerImagePair[imagePairIdx];
	__shared__ unsigned int basePtr;
	if (tidx == 0) {
		basePtr = atomicAdd(&d_globNumImagePairs[0], numMatches);
	}
	__syncthreads();


	if (tidx < numMatches) {
		const unsigned int srcAddr = imagePairIdx*MAX_MATCHES_PER_IMAGE_PAIR_FILTERED + tidx;

		uint2 currFilteredMachtKeyPointIndices = d_currFilteredMatchKeyPointIndices[srcAddr];


		const SIFTKeyPoint& k_i = d_keyPoints[currFilteredMachtKeyPointIndices.x];
		const SIFTKeyPoint& k_j = d_keyPoints[currFilteredMachtKeyPointIndices.y];

		EntryJ e;
		const unsigned int imageIdx0 = imagePairIdx;
		const unsigned int imageIdx1 = curFrame;
		e.imgIdx_i = imageIdx0;
		e.imgIdx_j = imageIdx1;
		e.pos_i = colorIntrinsicsInv * (k_i.depth * make_float3(k_i.pos.x, k_i.pos.y, 1.0f));
		e.pos_j = colorIntrinsicsInv * (k_j.depth * make_float3(k_j.pos.x, k_j.pos.y, 1.0f));

		d_globMatches[basePtr + tidx] = e;
		d_globMatchesKeyPointIndices[basePtr + tidx] = currFilteredMachtKeyPointIndices;
	}
}

void SIFTImageManager::AddCurrToResidualsCU(unsigned int curFrame, unsigned int startFrame, unsigned int numFrames, const float4x4& colorIntrinsicsInv) {
	if (numFrames == 0) return;

	dim3 grid(numFrames - startFrame);
	const unsigned int threadsPerBlock = ((MAX_MATCHES_PER_IMAGE_PAIR_FILTERED + 31) / 32) * 32;
	dim3 block(threadsPerBlock);

	if (m_timer) m_timer->startEvent(__FUNCTION__);

	AddCurrToResidualsCU_Kernel << <grid, block >> >(
		curFrame,
		startFrame,
		d_globMatches,
		d_globMatchesKeyPointIndices,
		d_globNumResiduals,
		d_currNumFilteredMatchesPerImagePair,
		d_currFilteredMatchKeyPointIndices,
		d_keyPoints,
		m_maxKeyPointsPerImage,
		colorIntrinsicsInv
		);

	cutilSafeCall(hipMemcpy(&m_globNumResiduals, d_globNumResiduals, sizeof(unsigned int), hipMemcpyDeviceToHost));

	if (m_timer) m_timer->endEvent();

	CheckErrorCUDA(__FUNCTION__);
}


#define INVALIDATEIMAGE_TO_IMAGE_KERNEL_THREADS_X 128

void __global__ InvalidateImageToImageCU_Kernel(EntryJ* d_globMatches, unsigned int globNumResiduals, uint2 imageToImageIdx)
{
	const unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;

	if (idx < globNumResiduals) {
		if (d_globMatches[idx].imgIdx_i == imageToImageIdx.x &&
			d_globMatches[idx].imgIdx_j == imageToImageIdx.y) {
			d_globMatches[idx].setInvalid();
		}

	}

}

void SIFTImageManager::InvalidateImageToImageCU(const uint2& imageToImageIdx) {

	const unsigned int threadsPerBlock = INVALIDATEIMAGE_TO_IMAGE_KERNEL_THREADS_X;
	dim3 grid((m_globNumResiduals + threadsPerBlock - 1) / threadsPerBlock);
	dim3 block(threadsPerBlock);

	if (m_timer) m_timer->startEvent(__FUNCTION__);

	InvalidateImageToImageCU_Kernel << <grid, block >> >(d_globMatches, m_globNumResiduals, imageToImageIdx);

	if (m_timer) m_timer->endEvent();

	CheckErrorCUDA(__FUNCTION__);
}


#define CHECK_FOR_INVALID_FRAMES_X 128
#define CHECK_FOR_INVALID_FRAMES_THREADS_X 16

void __global__ CheckForInvalidFramesCU_Kernel(const int* d_varToCorrNumEntriesPerRow, int* d_validImages, unsigned int numVars,
	EntryJ* d_globMatches, unsigned int numGlobResiduals)
{
	const unsigned int resIdx = blockDim.x*blockIdx.x + blockIdx.y;
	const unsigned int varIdx = gridDim.x*threadIdx.x + threadIdx.y;

	if (varIdx < numVars && resIdx < numGlobResiduals) {
		if (d_varToCorrNumEntriesPerRow[varIdx] == 0) {
			if (d_globMatches[resIdx].isValid() && (d_globMatches[resIdx].imgIdx_i == varIdx || d_globMatches[resIdx].imgIdx_j == varIdx)) {
				d_globMatches[resIdx].setInvalid();
			}
			if (d_validImages[varIdx] != 0) {
				if (varIdx == 0) printf("ERROR ERROR INVALIDATING THE FIRST FRAME\n");
				d_validImages[varIdx] = 0;
			}
		}
	}

}

void SIFTImageManager::CheckForInvalidFramesCU(const int* d_varToCorrNumEntriesPerRow, unsigned int numVars)
{
	dim3 block((m_globNumResiduals + CHECK_FOR_INVALID_FRAMES_X - 1) / CHECK_FOR_INVALID_FRAMES_X, CHECK_FOR_INVALID_FRAMES_X);
	dim3 threadsPerBlock((numVars + CHECK_FOR_INVALID_FRAMES_THREADS_X - 1) / CHECK_FOR_INVALID_FRAMES_THREADS_X, CHECK_FOR_INVALID_FRAMES_THREADS_X);

	if (m_timer) m_timer->startEvent(__FUNCTION__);

	cutilSafeCall(hipMemcpy(d_validImages, m_validImages.data(), sizeof(int) * numVars, hipMemcpyHostToDevice));

	CheckForInvalidFramesCU_Kernel << <block, threadsPerBlock >> >(d_varToCorrNumEntriesPerRow, d_validImages, numVars, d_globMatches, m_globNumResiduals);

	cutilSafeCall(hipMemcpy(m_validImages.data(), d_validImages, sizeof(int) * numVars, hipMemcpyDeviceToHost));

	if (m_timer) m_timer->endEvent();

	CheckErrorCUDA(__FUNCTION__);
}

void __global__ CheckForInvalidFramesSimpleCU_Kernel(const int* d_varToCorrNumEntriesPerRow, int* d_validImages, unsigned int numVars)
{
	const unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;

	if (idx < numVars) {
		if (d_varToCorrNumEntriesPerRow[idx] == 0) {
			d_validImages[idx] = 0;
		}
	}
}

void SIFTImageManager::CheckForInvalidFramesSimpleCU(const int* d_varToCorrNumEntriesPerRow, unsigned int numVars)
{
	const unsigned int threadsPerBlock = CHECK_FOR_INVALID_FRAMES_THREADS_X;
	dim3 grid((numVars + threadsPerBlock - 1) / threadsPerBlock);
	dim3 block(threadsPerBlock);

	if (m_timer) m_timer->startEvent(__FUNCTION__);

	cutilSafeCall(hipMemcpy(d_validImages, m_validImages.data(), sizeof(int) * numVars, hipMemcpyHostToDevice));

	CheckForInvalidFramesSimpleCU_Kernel << <grid, block >> >(d_varToCorrNumEntriesPerRow, d_validImages, numVars);

	cutilSafeCall(hipMemcpy(m_validImages.data(), d_validImages, sizeof(int) * numVars, hipMemcpyDeviceToHost));

	if (m_timer) m_timer->endEvent();

	CheckErrorCUDA(__FUNCTION__);
}


void __global__ TestSVDDebugCU_Kernel(float3x3* d_m, float3x3* d_u, float3x3* d_s, float3x3* d_v) {

	float3x3 m = d_m[0];

}





void SIFTImageManager::TestSVDDebugCU(const float3x3& m) {

	dim3 grid(1);
	dim3 block(1);

	float3x3* d_m, *d_u, *d_s, *d_v;
	cutilSafeCall(hipMalloc(&d_m, sizeof(float3x3)));
	cutilSafeCall(hipMalloc(&d_u, sizeof(float3x3)));
	cutilSafeCall(hipMalloc(&d_s, sizeof(float3x3)));
	cutilSafeCall(hipMalloc(&d_v, sizeof(float3x3)));


	cutilSafeCall(hipMemcpy(d_m, &m, sizeof(float3x3), hipMemcpyHostToDevice));

	CUDATimer timer;
	timer.startEvent(__FUNCTION__);

	TestSVDDebugCU_Kernel << <grid, block >> >(d_m, d_u, d_s, d_v);

	timer.endEvent();
	timer.evaluate();

	float3x3 u, s, v;
	cutilSafeCall(hipMemcpy(&u, d_u, sizeof(float3x3), hipMemcpyDeviceToHost));
	cutilSafeCall(hipMemcpy(&s, d_s, sizeof(float3x3), hipMemcpyDeviceToHost));
	cutilSafeCall(hipMemcpy(&v, d_v, sizeof(float3x3), hipMemcpyDeviceToHost));

	float3x3 res = u * s * v.getTranspose();
	res.print();
	printf("\n\n");

	CheckErrorCUDA(__FUNCTION__);

}



